#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <time.h>
#include <iostream>
#include <chrono>



__global__
    void addKernel(double* x, double* y, double* C, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n * n; i += stride) {
        int row = i / n;
        int col = i % n;

        C[row * n + col] = 0;
        for (int k = 0; k < n; k++) {
            C[row * n + col] += x[row * n + k] * y[k * n + col];
        }
    }
}

__global__
    void addKernel2(double* x, double* y, int n) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < n) {
        y[threadID] = x[threadID] + y[threadID];
    }
}
extern "C"
    void* addVectorsGPU(double* a, double* b, int n) {
    hipDeviceSynchronize();

    double* x;
    double* c;

    hipMalloc(&x, n * n * sizeof(double));
    hipMemcpy(x, a, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void**)&c, n * n * sizeof(double));
    //double* y=x;

    addKernel << < 1024, 1024 >> > (x, x, c, n);


    hipMemcpy(b, c, n * n * sizeof(double), hipMemcpyDeviceToHost);

    return b;
}


extern "C"
    void* addVectorsGPU2(double* a, double* b, int n) {
    hipDeviceSynchronize();

    double* x;

    hipMalloc(&x, n * sizeof(double));
    hipMemcpy(x, a, n * sizeof(double), hipMemcpyHostToDevice);
    double* y=x;

    addKernel2 << < (n - 1) / 1024 + 1, 1024 >> > (x, y, n);


    hipMemcpy(b, y, n * sizeof(double), hipMemcpyDeviceToHost);

    return b;
}
