#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "MMWithLimitCuda.cuh"
#include <iostream>
#include <chrono>
#include <cmath>
#include <time.h>


//YOU CAN SEE YOUR SYSTEM INDEX PROPERTY FOR THE CUDA SETTING BY THIS LINK:
//C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.3\extras\demo_suite\deviceQuery.exe






__global__ void vectorAdditionKernel(double* A, double* B, double* C, int n) {
   
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n * n; i += stride) {
        int row = i / n;
        int col = i % n;

        C[row * n + col] = 0;
        for (int k = 0; k < n; k++) {
            C[row * n + col] += A[row * n + k] * B[k * n + col];
        }
    }
}

/**
 * Wrapper function for the CUDA kernel function.
 * @param A Array A.
 * @param B Array B.
 * @param C Sum of array elements A and B directly across.
 * @param arraySize Size of arrays A, B, and C.
 */


void kernel(double* A, double* B, double* C, int n) {

    double* a, * b, * c11;
    // Allocate device memory.
    hipMalloc((void**)&a, n * n * sizeof(double));
    hipMalloc((void**)&b, n * n * sizeof(double));
    hipMalloc((void**)&c11, n * n * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(a, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b, B, n * n * sizeof(double), hipMemcpyHostToDevice);


    int THREADS = 32;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    int BLOCKS = n / THREADS;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);


    vectorAdditionKernel << < 1024, 1024 >> > (a, b, c11, n);


    hipDeviceSynchronize();

    hipMemcpy(C, c11, n*n * sizeof(double), hipMemcpyDeviceToHost);

}


